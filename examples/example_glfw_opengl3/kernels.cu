#include "hip/hip_runtime.h"
#include <GLFW/glfw3.h>

#include <cuda_gl_interop.h>
#include <>
#include <hip/hip_runtime.h>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include <stdio.h>

#if defined(_MSC_VER) && (_MSC_VER >= 1900) && !defined(IMGUI_DISABLE_WIN32_FUNCTIONS)
#pragma comment(lib, "legacy_stdio_definitions")
#endif

__global__ void offset_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst, int width, int height, unsigned char offset)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	pixel.x += offset;
	pixel.y += offset;
	surf2Dwrite(pixel, dst, col * 4, row);
}

__global__ void commit_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst, int width, int height)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	surf2Dwrite(pixel, dst, col * 4, row);
}
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
__global__ void transpose_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst, int width, int height)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	surf2Dwrite(pixel, dst, row * 4, col);
}

GLFWwindow* window;
unsigned int original_texture;
unsigned int texture;
hipSurfaceObject_t original_surface;
hipSurfaceObject_t surface;
const size_t width = 1024;
const size_t height = 1024;

void init()
{
	float* image = new float[width * height * 4];
	for (size_t row = 0; row < height; ++row)
	{
		for (size_t col = 0; col < width; ++col)
		{
			image[(width * row + col) * 4 + 0] = (row % 32) / 32.0f;
			image[(width * row + col) * 4 + 1] = (col % 32) / 32.0f;
			image[(width * row + col) * 4 + 2] = (float)row / width;
			image[(width * row + col) * 4 + 3] = 255;
		}
	}

	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_FLOAT, image);

	glGenTextures(1, &original_texture);
	glBindTexture(GL_TEXTURE_2D, original_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_FLOAT, image);

	{
		hipGraphicsResource* cuda_resource;
		hipGraphicsGLRegisterImage(&cuda_resource, (GLuint)texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
		hipGraphicsMapResources(1, &cuda_resource);

		hipArray* cuda_array;
		hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource, 0, 0);

		hipResourceDesc res_desc{};
		res_desc.resType = hipResourceTypeArray;
		res_desc.res.array.array = cuda_array;

		hipCreateSurfaceObject(&surface, &res_desc);
	}
	{
		hipGraphicsResource* cuda_resource;
		hipGraphicsGLRegisterImage(&cuda_resource, (GLuint)original_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
		hipGraphicsMapResources(1, &cuda_resource);

		hipArray* cuda_array;
		hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource, 0, 0);

		hipResourceDesc res_desc{};
		res_desc.resType = hipResourceTypeArray;
		res_desc.res.array.array = cuda_array;

		hipCreateSurfaceObject(&original_surface, &res_desc);
	}
}

void frame()
{
	ImGui::Begin("Main Window", 0, ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoMove);

	{
		ImGuiIO& io = ImGui::GetIO();

		ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);

		const size_t samples_count = 1000;
		static float samples[samples_count];
		static size_t values_offset = 0;
		samples[values_offset] = io.Framerate;
		values_offset += 1;
		values_offset %= 1000;
		float average = 0.0f;
		for (int n = 0; n < samples_count; n++)
			average += samples[n];
		average /= (float)samples_count;
		char overlay[32];
		sprintf(overlay, "avg %f", average);
		ImGui::PlotLines("FPS", samples, samples_count, values_offset, overlay, 500, 900, ImVec2(0, 80.0f));
	}

	{
		static bool offset_apply = true;
		static int offset = 0;
		int min = 0, max = 255;
		ImGui::Checkbox("Active", &offset_apply);
		ImGui::SameLine();
		ImGui::SliderScalar("Offset", ImGuiDataType_U8, &offset, &min, &max);
		if (offset_apply)
		{
			dim3 blockSize(32, 32);
			dim3 gridSize(width / 32, height / 32);
			offset_kernel<<<gridSize, blockSize>>> (original_surface, surface, width, height, offset);
			hipDeviceSynchronize();
		}

		if (ImGui::Button("Tranpose"))
		{
			dim3 blockSize(32, 32);
			dim3 gridSize(width / 32, height / 32);
			transpose_kernel<<<gridSize, blockSize>>>(original_surface, surface, width, height);
			hipDeviceSynchronize();
		}

		if (ImGui::Button(">>>	"))
		{
			dim3 blockSize(32, 32);
			dim3 gridSize(width / 32, height / 32);
			commit_kernel<<<gridSize, blockSize>>>(original_surface, surface, width, height);
			hipDeviceSynchronize();

		}
		ImGui::SameLine();
		if (ImGui::Button("<<<"))
		{
			dim3 blockSize(32, 32);
			dim3 gridSize(width / 32, height / 32);
			commit_kernel <<<gridSize, blockSize>>>(surface, original_surface, width, height);
			hipDeviceSynchronize();
		}
		float image_height = ImGui::GetWindowHeight() - ImGui::GetCursorPosY() - 30;
		ImGui::Image((void*)original_texture, { image_height, image_height });
		ImGui::SameLine();
		ImGui::Image((void*)texture, { image_height, image_height });
	}

	ImGui::End();

	static bool show_imgui_demo_window = false;
	if (ImGui::IsKeyPressed(ImGuiKey_F1))
		show_imgui_demo_window = !show_imgui_demo_window;
	if (show_imgui_demo_window)
		ImGui::ShowDemoWindow(&show_imgui_demo_window);
}

int main(int, char**)
{
	glfwInit();
	const char* glsl_version = "#version 130";
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
	glfwWindowHint(GLFW_MAXIMIZED, 1);
	window = glfwCreateWindow(1280, 720, "Dear ImGui GLFW+OpenGL3 example", nullptr, nullptr);
	glfwMakeContextCurrent(window);
	glfwSwapInterval(0);

	ImGui::CreateContext();

	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init(glsl_version);

	init();

	ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);

	while (!glfwWindowShouldClose(window))
	{
		glfwPollEvents();
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		int window_width, window_height;
		glfwGetWindowSize(window, &window_width, &window_height);
		ImGui::GetWindowWidth();
		ImGui::SetNextWindowPos({ 0, 0 });
		ImGui::SetNextWindowSize({ (float)window_width, (float)window_height });

		frame();

		ImGui::Render();
		int display_w, display_h;
		glfwGetFramebufferSize(window, &display_w, &display_h);
		glViewport(0, 0, display_w, display_h);
		glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w, clear_color.w);
		glClear(GL_COLOR_BUFFER_BIT);
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		glfwSwapBuffers(window);
	}

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();

	glfwDestroyWindow(window);
	glfwTerminate();

	return 0;
}

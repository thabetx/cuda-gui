#include "hip/hip_runtime.h"
#include <GLFW/glfw3.h>

#include <cuda_gl_interop.h>
#include <>
#include <hip/hip_runtime.h>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#if defined(_MSC_VER) && (_MSC_VER >= 1900) && !defined(IMGUI_DISABLE_WIN32_FUNCTIONS)
#pragma comment(lib, "legacy_stdio_definitions")
#endif

__global__ void copy_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	surf2Dwrite(pixel, dst, col * 4, row);
}

__global__ void offset_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst, int offset)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	pixel.x += offset;
	pixel.y += offset;
	surf2Dwrite(pixel, dst, col * 4, row);
}

__global__ void transpose_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	uchar4 pixel;
	surf2Dread(&pixel, src, col * 4, row);
	surf2Dwrite(pixel, dst, row * 4, col);
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
__global__ void transpose_coalesced_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst)
{
	__shared__ uchar4 data[TILE_DIM + 1][TILE_DIM + 1];
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	uchar4 pixel;
	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
	{
		surf2Dread(&pixel, src, x * 4, y+j);
		data[threadIdx.y + j][threadIdx.x] = pixel;
	}

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
	{
		surf2Dwrite(data[threadIdx.x][threadIdx.y + j], dst, x * 4, y+j);
	}
}

__global__ void blur_x_kernel(hipSurfaceObject_t src, hipSurfaceObject_t dst)
{
}

GLFWwindow* window;
unsigned int original_texture;
unsigned int texture;
hipSurfaceObject_t original_surface;
hipSurfaceObject_t surface;
const size_t width = 2048;
const size_t height = 2048;
hipEvent_t start_event, end_event;
float elapsed_time_ms;
const size_t kernel_rounds = 50;


void init()
{
	int image_width = 0;
	int image_height = 0;
	unsigned char* image = stbi_load("data\\nick-fewings-u4QnZJB4sT0-unsplash.jpg", &image_width, &image_height, NULL, 4);
	if (image == nullptr || image_width != width || image_height != height)
	{
		stbi_image_free(image);
		image = new unsigned char[width * height * 4];
		for (size_t row = 0; row < height; ++row)
		{
			for (size_t col = 0; col < width; ++col)
			{
				image[(width * row + col) * 4 + 0] = (row % 255);
				image[(width * row + col) * 4 + 1] = (col % 255);
				image[(width * row + col) * 4 + 2] = ((row+col) % 255);
				image[(width * row + col) * 4 + 3] = 255;
			}
		}
	}

	glGenTextures(1, &original_texture);
	glBindTexture(GL_TEXTURE_2D, original_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, image);

	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, image);

	{
		hipGraphicsResource* cuda_resource;
		hipGraphicsGLRegisterImage(&cuda_resource, (GLuint)original_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
		hipGraphicsMapResources(1, &cuda_resource);

		hipArray* cuda_array;
		hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource, 0, 0);

		hipResourceDesc res_desc{};
		res_desc.resType = hipResourceTypeArray;
		res_desc.res.array.array = cuda_array;

		hipCreateSurfaceObject(&original_surface, &res_desc);
	}
	{
		hipGraphicsResource* cuda_resource;
		hipGraphicsGLRegisterImage(&cuda_resource, (GLuint)texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
		hipGraphicsMapResources(1, &cuda_resource);

		hipArray* cuda_array;
		hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource, 0, 0);

		hipResourceDesc res_desc{};
		res_desc.resType = hipResourceTypeArray;
		res_desc.res.array.array = cuda_array;

		hipCreateSurfaceObject(&surface, &res_desc);
	}

	{
		hipEventCreate(&start_event);
		hipEventCreate(&end_event);
	}
}

void frame()
{
	ImGui::Begin("Main Window", 0, ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoMove);

	{
		ImGui::BeginChild("Child Left", ImVec2(ImGui::GetContentRegionAvail().x * 0.5f, 260));

		static enum Kernel {
			none,
			copy,
			copy_back,
			shift,
			tranpose,
			tranpose_coalesced,
			blur,
		} kernel = blur;

		ImGui::RadioButton("None", (int*)&kernel, none);
		ImGui::RadioButton("Copy >>>", (int*)&kernel, copy);
		ImGui::RadioButton("Copy <<<", (int*)&kernel, copy_back);
		ImGui::RadioButton("shift", (int*)&kernel, shift);
		ImGui::RadioButton("Transpose", (int*)&kernel, tranpose);
		ImGui::RadioButton("Transpose Coalesced", (int*)&kernel, tranpose_coalesced);
		ImGui::RadioButton("Blur", (int*)&kernel, blur);

		if (ImGui::IsKeyPressed(ImGuiKey_1)) kernel = Kernel(0);
		if (ImGui::IsKeyPressed(ImGuiKey_2)) kernel = Kernel(1);
		if (ImGui::IsKeyPressed(ImGuiKey_3)) kernel = Kernel(2);
		if (ImGui::IsKeyPressed(ImGuiKey_4)) kernel = Kernel(3);
		if (ImGui::IsKeyPressed(ImGuiKey_5)) kernel = Kernel(4);
		if (ImGui::IsKeyPressed(ImGuiKey_6)) kernel = Kernel(5);
		if (ImGui::IsKeyPressed(ImGuiKey_7)) kernel = Kernel(6);

		static Kernel previous_kernel;
		if (ImGui::IsKeyPressed(ImGuiKey_Tab, false)) {
			printf("nice\n");
			previous_kernel = kernel;
			kernel = copy;
		}
		if (ImGui::IsKeyReleased(ImGuiKey_Tab)) {
			printf("done\n");
			kernel = previous_kernel;
		}

		dim3 grid_size, block_size;
		static int offset = 0;
		static int blur_rounds = 1;

		if (kernel == shift)
		{
			int min = 0, max = 255;
			ImGui::SliderScalar("Offset", ImGuiDataType_U8, &offset, &min, &max);
			block_size = dim3(32, 32);
			grid_size = dim3(width / 32, height / 32);
		}

		if (kernel == blur)
		{
			ImGui::SliderInt("Blur Rounds", &blur_rounds, 0, 20);
		}

		hipEventRecord(start_event, 0);

		for (size_t i = 0; i < kernel_rounds; ++i)
		{
			switch (kernel)
			{
			case none:
				break;
			case copy:
			{
				block_size = dim3(32, 32);
				grid_size = dim3(width / 32, height / 32);
				copy_kernel << <grid_size, block_size >> > (original_surface, surface);
				break;
			}
			case copy_back:
			{
				block_size = dim3(32, 32);
				grid_size = dim3(width / 32, height / 32);
				copy_kernel << <grid_size, block_size >> > (surface, original_surface);
				break;
			}
			case shift:
			{
				block_size = dim3(32, 32);
				grid_size = dim3(width / 32, height / 32);
				offset_kernel << <grid_size, block_size >> > (original_surface, surface, offset);
				break;
			}
			case tranpose:
			{
				block_size = dim3(32, 32);
				grid_size = dim3(width / 32, height / 32);
				transpose_kernel << <grid_size, block_size >> > (original_surface, surface);
				break;
			}
			case tranpose_coalesced:
			{
				block_size = dim3(TILE_DIM, BLOCK_ROWS);
				grid_size = dim3(width / TILE_DIM, height / TILE_DIM);
				transpose_coalesced_kernel << <grid_size, block_size >> > (original_surface, surface);
				break;
			}
			case blur:
			{
				block_size = dim3(1, 1);
				grid_size = dim3(1, height);
				blur_x_kernel << <grid_size, block_size >> > (original_surface, surface);
				break;
			}
			default:
				break;
			}
		}
		hipEventRecord(end_event, 0);
		hipEventSynchronize(end_event);
		hipEventElapsedTime(&elapsed_time_ms, start_event, end_event);
		hipDeviceSynchronize();

		ImGui::EndChild();
	}

	ImGui::SameLine();

	{
		ImGui::BeginChild("Child Right", ImVec2(0, 260));

		ImGuiIO& io = ImGui::GetIO();
		ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);

		const size_t samples_count = 100;
		static float samples[samples_count];
		static size_t values_offset = 0;
		samples[values_offset] = elapsed_time_ms*1000/kernel_rounds;
		values_offset += 1;
		values_offset %= samples_count;
		float average = 0.0f;
		for (int n = 0; n < samples_count; n++)
			average += samples[n];
		average /= (float)samples_count;
		char overlay[32];
		sprintf(overlay, "avg %.0fus", average);
		ImGui::PlotLines("Kernel Time", samples, samples_count, values_offset, overlay, 0, 1000, ImGui::GetContentRegionAvail());
		ImGui::EndChild();
	}

	{
		float image_height = ImGui::GetWindowHeight() - ImGui::GetCursorPosY() - 30;
		ImGui::Image((void*)original_texture, { image_height, image_height }, { 0,0 }, { 1,1 }, { 1,1,1,1 }, { 1,1,1,1 });
		ImGui::SameLine();
		ImGui::Image((void*)texture, { image_height, image_height }, { 0,0 }, { 1,1 }, { 1,1,1,1 }, { 1,1,1,1 });
	}

	ImGui::End();

	static bool show_imgui_demo_window = false;
	if (ImGui::IsKeyPressed(ImGuiKey_F1))
		show_imgui_demo_window = !show_imgui_demo_window;
	if (show_imgui_demo_window)
		ImGui::ShowDemoWindow(&show_imgui_demo_window);
}

int main(int, char**)
{
	glfwInit();
	const char* glsl_version = "#version 130";
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
	glfwWindowHint(GLFW_MAXIMIZED, 1);
	window = glfwCreateWindow(1280, 720, "Dear ImGui GLFW+OpenGL3 example", nullptr, nullptr);
	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);

	ImGui::CreateContext();

	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init(glsl_version);

	init();

	ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);

	while (!glfwWindowShouldClose(window))
	{
		glfwPollEvents();
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		int window_width, window_height;
		glfwGetWindowSize(window, &window_width, &window_height);
		ImGui::GetWindowWidth();
		ImGui::SetNextWindowPos({ 0, 0 });
		ImGui::SetNextWindowSize({ (float)window_width, (float)window_height });

		frame();

		ImGui::Render();
		int display_w, display_h;
		glfwGetFramebufferSize(window, &display_w, &display_h);
		glViewport(0, 0, display_w, display_h);
		glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w, clear_color.w);
		glClear(GL_COLOR_BUFFER_BIT);
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		glfwSwapBuffers(window);
	}

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();

	glfwDestroyWindow(window);
	glfwTerminate();

	return 0;
}
